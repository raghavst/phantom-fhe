#include "hip/hip_runtime.h"
#include "bench_utils.cuh"
#include "util.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

void ckks_performance_test(EncryptionParameters &parms, double scale) {
    PhantomContext context(parms);
    print_parameters(context);
    cout << endl;

    cuda_stream_wrapper stream;

    print_timer_banner();

    auto count = 100;

    {
        CUDATimer timer("gen_secretkey");
        for (auto i = 0; i < count; i++) {
            timer.start();
            PhantomSecretKey secret_key(context);
            timer.stop();
        }
    }

    PhantomSecretKey secret_key(context);

    {
        CUDATimer timer("gen_publickey");
        for (auto i = 0; i < count; i++) {
            timer.start();
            PhantomPublicKey public_key = secret_key.gen_publickey(context);
            timer.stop();
        }
    }

    PhantomPublicKey public_key = secret_key.gen_publickey(context);

    // Generate relinearization keys
    {
        CUDATimer timer("gen_relinkey");
        for (auto i = 0; i < count; i++) {
            timer.start();
            PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
            timer.stop();
        }
    }

    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);

    PhantomGaloisKey gal_keys = secret_key.create_galois_keys(context);

    PhantomCKKSEncoder ckks_encoder(context);

    /*
    Populate a vector of floating-point values to batch.
    */
    std::vector<hipDoubleComplex> x;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);
    for (size_t i = 0; i < ckks_encoder.slot_count(); i++) {
        x.push_back(make_hipDoubleComplex(dis(gen), dis(gen)));
    }

    /*
    [Encoding]
    For scale we use the square root of the last coeff_modulus prime
    from parms.
    */
    PhantomPlaintext plain;
    {
        CUDATimer timer("encode");
        for (auto i = 0; i < count; i++) {
            timer.start();
            ckks_encoder.encode(context, x, scale, plain, 1);
            timer.stop();
        }
    }

    /*
    [Decoding]
    */
    {
        CUDATimer timer("decode");
        for (auto i = 0; i < count; i++) {
            timer.start();
            auto pod_vector2 = ckks_encoder.decode<hipDoubleComplex>(context, plain);
            timer.stop();
        }
    }

    /*
    [Encryption]
    */
    PhantomCiphertext encrypted;
    {
        CUDATimer timer("encrypt_asymmetric");
        for (auto i = 0; i < count; i++) {
            timer.start();
            public_key.encrypt_asymmetric(context, plain, encrypted);
            timer.stop();
        }
    }

    /*
    [Decryption]
    */
    PhantomPlaintext plain2;
    {
        CUDATimer timer("decrypt");
        for (auto i = 0; i < count; i++) {
            timer.start();
            secret_key.decrypt(context, encrypted, plain2);
            timer.stop();
        }
    }

    // homomorphic operations
    std::vector<hipDoubleComplex> pod_vector3(ckks_encoder.slot_count());
    std::vector<hipDoubleComplex> pod_vector4(ckks_encoder.slot_count());

    PhantomCiphertext encrypted1;
    for (size_t j = 0; j < ckks_encoder.slot_count(); j++)
        pod_vector3[j] = make_hipDoubleComplex(double(1), double(0));
    ckks_encoder.encode(context, pod_vector3, scale, plain, 1);
    public_key.encrypt_asymmetric(context, plain, encrypted1);

    PhantomCiphertext encrypted2;
    for (size_t j = 0; j < ckks_encoder.slot_count(); j++)
        pod_vector4[j] = make_hipDoubleComplex(double(1), double(0));
    ckks_encoder.encode(context, pod_vector4, scale, plain2, 1);
    public_key.encrypt_asymmetric(context, plain2, encrypted2);

    /*
    [Add]
    */
    {
        CUDATimer timer("add");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            add_inplace(context, tmp_ct, encrypted2);
            timer.stop();
        }
    }

    /*
    [Multiply]
    */
    {
        CUDATimer timer("multiply");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            multiply_inplace(context, tmp_ct, encrypted2);
            relinearize_inplace(context, tmp_ct, relin_keys);
            timer.stop();
        }
    }

    /*
    [Rescale]
    */
    {
        CUDATimer timer("rescale_to_next");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            multiply_inplace(context, tmp_ct, encrypted2);
            relinearize_inplace(context, tmp_ct, relin_keys);
            timer.start();
            rescale_to_next_inplace(context, tmp_ct);
            timer.stop();
        }
    }

    /*
    [Rotate Vector]
    */
    {
        CUDATimer timer("rotate_vector_one_step");
        for (auto i = 0; i < count; i++) {
            PhantomCiphertext tmp_ct(encrypted1);
            timer.start();
            rotate_inplace(context, tmp_ct, 1, gal_keys);
            timer.stop();
        }
    }
}

int main() {
    print_example_banner("CKKS Performance Test with Degree: 65536");
    std::vector<int> galois_steps = {1};
    {
        EncryptionParameters parms(scheme_type::ckks);
        size_t poly_modulus_degree = 1 << 16;
        parms.set_poly_modulus_degree(poly_modulus_degree);
        parms.set_galois_elts(get_elts_from_steps(galois_steps, poly_modulus_degree));
        parms.set_coeff_modulus(CoeffModulus::Create(
                poly_modulus_degree, {60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 
                                      60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60}));
        parms.set_special_modulus_size(6);
        double scale = pow(2.0, 40);
        ckks_performance_test(parms, scale);
    }

    return 0;
}
